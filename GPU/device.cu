#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

#define material 0.2545

__global__
void heat(const unsigned char* const in_world,
                     unsigned char* const out_world,
                     unsigned*      const heaters,
                             unsigned int numRows,
                             unsigned int numCols,
                             unsigned int numOfHeaters)
{

    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                          blockIdx.y * blockDim.y + threadIdx.y);
    // Check for out of boundries
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;

    const unsigned int thread_1D_pos  = thread_2D_pos.y * numCols + thread_2D_pos.x;
    // DEBUG
    /*if (thread_1D_pos == 0)
    {
        for (unsigned idx = 0; idx < numOfHeaters; idx++)
        {
          printf("Heater ID: %d\n", heaters[idx] ); 
          printf("\n\n\n"); 
        }

    }*/
    // DEBUG



    float current = (float)in_world[thread_1D_pos];
    float top     = (float)in_world[(thread_1D_pos-numCols)%(numRows*numCols)];
    float bottom  = (float)in_world[(thread_1D_pos+numCols)%(numRows*numCols)];
    float left    = (float)in_world[(thread_1D_pos-1)      %(numRows*numCols)];
    float right   = (float)in_world[(thread_1D_pos+1)      %(numRows*numCols)];

    float resultFloat = current+material*(top+bottom+left+right-4*current);

    if (resultFloat > 255.0)
    {
      out_world[thread_1D_pos] = 255;
    }
    else
    {
      out_world[thread_1D_pos] = (unsigned char)resultFloat;
    }

    for (unsigned idx = 0; idx < numOfHeaters; idx++)
    {
      if(heaters[idx] == thread_1D_pos){
        out_world[thread_1D_pos] = 255;
      }
    }

}

void heat_cuda(unsigned char* const d_inWorld,
               unsigned char* const d_outWorld,
               unsigned*      const d_heaters,
                              const size_t numRows,
                              const size_t numCols,
                              const size_t numOfHeaters)
{
    const dim3 blockSize ( 32,32 );

                              // Ceiling
    const dim3 gridSize ( 1 + ((numCols - 1) / blockSize.x),   1 + ((numRows - 1) / blockSize.y) );
    //const dim3 gridSize ( numCols,numRows,1 );
    // Launch a kernel
    heat <<<gridSize, blockSize>>>(d_inWorld, d_outWorld, d_heaters, numRows, numCols, numOfHeaters);

    // Make sure I didn't did not mess this up like I did with my last relationship
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
